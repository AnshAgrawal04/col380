#include "hip/hip_runtime.h"
#include "header.h"
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <vector>
#include <unordered_map>
#include <algorithm>
#include <cstring>                       

__global__
void bcsr_mul_kernel(int nb_rows, int k,
                     const int* __restrict__ A_rowptr,
                     const int* __restrict__ A_colind,
                     const unsigned long long* __restrict__ A_vals,
                     const int* __restrict__ B_rowptr,
                     const int* __restrict__ B_colind,
                     const unsigned long long* __restrict__ B_vals,
                     const int* __restrict__ C_rowptr,
                     const int* __restrict__ C_colind,
                     unsigned long long* C_vals)
{
    int i   = blockIdx.x;            
    int tid = threadIdx.x; 
    int elems = k * k;

    extern __shared__ unsigned long long shmem[];
    unsigned long long* Ash = shmem;
    unsigned long long* Bsh = shmem + elems;

    int bi = tid / k;
    int bj = tid % k;

    for (int a = A_rowptr[i]; a < A_rowptr[i+1]; ++a) {
        int p = A_colind[a];
        const unsigned long long* Ablk = A_vals + size_t(a) * elems;
        Ash[tid] = Ablk[tid];
        __syncthreads();

        for (int b = B_rowptr[p]; b < B_rowptr[p+1]; ++b) {
            int j = B_colind[b];
            const unsigned long long* Bblk = B_vals + size_t(b) * elems;
            Bsh[tid] = Bblk[tid];
            __syncthreads();

            unsigned long long sum = 0;
            for (int kk = 0; kk < k; ++kk) {
                sum += Ash[bi*k + kk] * Bsh[kk*k + bj];
            }

            int low = C_rowptr[i];
            int high = C_rowptr[i+1] - 1;
            while (low <= high) {
                int mid = (low + high) >> 1;
                int col = C_colind[mid];
                if (col == j) { low = mid; break; }
                if (col < j) low = mid + 1;
                else         high = mid - 1;
            }
            int pos = low;

            // Atomic accumulate
            atomicAdd(&C_vals[size_t(pos)*elems + bi*k + bj], sum);
            __syncthreads();
        }
    }
}

void CSR_mul(struct BCSR& A, struct BCSR& B, struct BCSR& C, int k) {

    int nb_rows = (A.rows+(k-1)) / k;
    vector<std::unordered_map<int,bool>> rowBlocks(nb_rows);
    for (int i = 0; i < nb_rows; ++i) {
        for (int a = A.rowptr[i]; a < A.rowptr[i+1]; ++a) {
            int p = A.colind[a];
            for (int b = B.rowptr[p]; b < B.rowptr[p+1]; ++b) {
                rowBlocks[i][B.colind[b]] = true;
            }
        }
    }

    C.rows = A.rows;
    C.cols = B.cols;    
    C.rowptr  = new int[nb_rows + 1];
    C.rowptr[0]  = 0;


    for (int i = 0; i < nb_rows; ++i)
        C.rowptr[i+1] = C.rowptr[i] + int(rowBlocks[i].size());


    int nnzC = C.rowptr[nb_rows];

    C.nz = nnzC;
    // cout<<"finally c nnz "<<nnzC<<endl;
    C.colind = new int[nnzC];
    int idx = 0;

    for (int i = 0; i < nb_rows; ++i) {
        std::vector<int> cols;
        cols.reserve(rowBlocks[i].size());
        for (auto &kv : rowBlocks[i]) cols.push_back(kv.first);
        std::sort(cols.begin(), cols.end());
        for (int col : cols) C.colind[idx++] = col;
    }

    // Allocate and zero C.values
    size_t elems = size_t(k) * k;
    C.values = new unsigned long long[nnzC * elems];
    std::memset(C.values, 0, sizeof(unsigned long long) * nnzC * elems);

    // Device buffers
    int *d_A_rowptr, *d_A_colind, *d_B_rowptr, *d_B_colind, *d_C_rowptr, *d_C_colind;
    unsigned long long *d_A_vals, *d_B_vals, *d_C_vals;

    // Helper: alloc+copy
    auto alloc_and_copy = [&](const void* src, size_t bytes, void** dst) {
        hipMalloc(dst, bytes);
        hipMemcpy(*dst, src, bytes, hipMemcpyHostToDevice);
    };
    int padded_b = (B.rows + k -1)/k ;
    alloc_and_copy(A.rowptr, (nb_rows+1)*sizeof(int), (void**)&d_A_rowptr);
    alloc_and_copy(A.colind, A.rowptr[nb_rows]*sizeof(int), (void**)&d_A_colind);
    alloc_and_copy(A.values, elems*A.rowptr[nb_rows]*sizeof(unsigned long long), (void**)&d_A_vals);
    alloc_and_copy(B.rowptr, (padded_b+1)*sizeof(int),   (void**)&d_B_rowptr);
    alloc_and_copy(B.colind, B.rowptr[padded_b]*sizeof(int), (void**)&d_B_colind);
    alloc_and_copy(B.values, elems*B.rowptr[padded_b]*sizeof(unsigned long long), (void**)&d_B_vals);
    alloc_and_copy(C.rowptr, (nb_rows+1)*sizeof(int),  (void**)&d_C_rowptr);
    alloc_and_copy(C.colind, nnzC*sizeof(int),  (void**)&d_C_colind);

    hipMalloc((void**)&d_C_vals, elems*nnzC*sizeof(unsigned long long));
    hipMemset(d_C_vals, 0, elems*nnzC*sizeof(unsigned long long));

    dim3 grid(nb_rows);
    dim3 block(k * k);
    size_t sharedMem = 2ULL * k * k * sizeof(unsigned long long);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess){
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
        cout<<"This one error"<<endl;
    }
        

    bcsr_mul_kernel<<<grid, block, sharedMem>>>(
        nb_rows, k,
        d_A_rowptr, d_A_colind, d_A_vals,
        d_B_rowptr, d_B_colind, d_B_vals,
        d_C_rowptr, d_C_colind, d_C_vals
    );
    err = hipGetLastError();
    if (err != hipSuccess)
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));

    hipDeviceSynchronize();

    hipMemcpy(C.values, d_C_vals, elems*nnzC*sizeof(unsigned long long), hipMemcpyDeviceToHost);

    hipFree(d_A_rowptr); hipFree(d_A_colind); hipFree(d_A_vals);
    hipFree(d_B_rowptr); hipFree(d_B_colind); hipFree(d_B_vals);
    hipFree(d_C_rowptr); hipFree(d_C_colind); hipFree(d_C_vals);
}